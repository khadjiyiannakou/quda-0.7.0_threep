#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string>
#include <iostream>

#include <color_spinor_field.h>
#include <clover_field.h>

// these control the Wilson-type actions
#ifdef GPU_WILSON_DIRAC
//#define DIRECT_ACCESS_LINK
//#define DIRECT_ACCESS_WILSON_SPINOR
//#define DIRECT_ACCESS_WILSON_ACCUM
//#define DIRECT_ACCESS_WILSON_INTER
//#define DIRECT_ACCESS_WILSON_PACK_SPINOR
//#define DIRECT_ACCESS_CLOVER
#endif // GPU_WILSON_DIRAC


#include <quda_internal.h>
#include <dslash_quda.h>
#include <sys/time.h>
#include <blas_quda.h>
#include <face_quda.h>

#include <inline_ptx.h>

namespace quda {

  namespace ndegtwisted {

#include <dslash_constants.h>
#include <dslash_textures.h>
#include <dslash_index.cuh>

    // Enable shared memory dslash for Fermi architecture
    //#define SHARED_WILSON_DSLASH
    //#define SHARED_8_BYTE_WORD_SIZE // 8-byte shared memory access

    //#if (__COMPUTE_CAPABILITY__) >= 200 && defined(GPU_NDEG_TWISTED_MASS_DIRAC)
#if (__COMPUTE_CAPABILITY__ >= 200) && defined(GPU_NDEG_TWISTED_MASS_DIRAC)
#include <tm_ndeg_dslash_def.h>   // Non-degenerate twisted Mass
#endif

#ifndef NDEGTM_SHARED_FLOATS_PER_THREAD
#define NDEGTM_SHARED_FLOATS_PER_THREAD 0
#endif

#include <dslash_quda.cuh>

  } // end namespace twisted
  
  // declare the dslash events
#include <dslash_events.cuh>

  using namespace ndegtwisted;

#if (__COMPUTE_CAPABILITY__ >= 200) && defined(GPU_NDEG_TWISTED_MASS_DIRAC)
  template <typename sFloat, typename gFloat>
  class NdegTwistedDslashCuda : public SharedDslashCuda {

  private:
    const gFloat *gauge0, *gauge1;
    const QudaTwistDslashType dslashType;
    double a, b, c, d;

  protected:
    unsigned int sharedBytesPerThread() const
    {
#if (__COMPUTE_CAPABILITY__ >= 200)
      if (dslashParam.kernel_type == INTERIOR_KERNEL) {
	int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
	return NDEGTM_SHARED_FLOATS_PER_THREAD * reg_size;
      } else {
	return 0;
      }
#else
      int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
      return NDEGTM_SHARED_FLOATS_PER_THREAD * reg_size;
#endif
    }

  public:
    NdegTwistedDslashCuda(cudaColorSpinorField *out, const gFloat *gauge0, const gFloat *gauge1, 
		      const QudaReconstructType reconstruct, const cudaColorSpinorField *in,  const cudaColorSpinorField *x, 
		      const QudaTwistDslashType dslashType, const double kappa, const double mu, 
		      const double epsilon, const double k, const int dagger)
      : SharedDslashCuda(out, in, x, reconstruct, dagger), gauge0(gauge0), gauge1(gauge1), dslashType(dslashType)
    { 
      bindSpinorTex<sFloat>(in, out, x); 
      a = kappa;
      b = mu;
      c = epsilon;
      d = k;
      if (dslashType != QUDA_NONDEG_DSLASH) errorQuda("Invalid dslashType for non-degenerate twisted-mass Dslash");
      dslashParam.fl_stride = in->VolumeCB()/2;
    }
    virtual ~NdegTwistedDslashCuda() { unbindSpinorTex<sFloat>(in, out, x); }

    TuneKey tuneKey() const
    {
      TuneKey key = DslashCuda::tuneKey();
      strcat(key.aux,",NdegDslash");
      return key;
    }

    void apply(const hipStream_t &stream)
    {

#ifdef SHARED_WILSON_DSLASH
      if (dslashParam.kernel_type == EXTERIOR_KERNEL_X) 
	errorQuda("Shared dslash does not yet support X-dimension partitioning");
#endif
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      NDEG_TM_DSLASH(twistedNdegMassDslash, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam,
		     (sFloat*)out->V(), (float*)out->Norm(), gauge0, gauge1, 
		     (sFloat*)in->V(), (float*)in->Norm(), a, b, c, d, (sFloat*)(x ? x->V() : 0), (float*)(x ? x->Norm() : 0));
    }

    long long flops() const { return (x ? 1416ll : 1392ll) * in->VolumeCB(); } // FIXME for multi-GPU
  };
#endif // GPU_NDEG_TWISTED_MASS_DIRAC


#include <dslash_policy.cuh> 

  void ndegTwistedMassDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, 
				 const cudaColorSpinorField *in, const int parity, const int dagger, 
				 const cudaColorSpinorField *x, const QudaTwistDslashType type, 
				 const double &kappa, const double &mu, const double &epsilon, 
				 const double &k,  const int *commOverride, TimeProfile &profile, 
				 const QudaDslashPolicy &dslashPolicy)
  {
    inSpinor = (cudaColorSpinorField*)in; // EVIL
#if (__COMPUTE_CAPABILITY__ >= 200) && defined(GPU_NDEG_TWISTED_MASS_DIRAC)
    int Npad = (in->Ncolor()*in->Nspin()*2)/in->FieldOrder(); // SPINOR_HOP in old code

    int ghost_threads[4] = {0};
    int bulk_threads = in->Volume() / 2;

    for(int i=0;i<4;i++){
      dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
      dslashParam.ghostOffset[i] = Npad*(in->GhostOffset(i) + in->Stride());
      dslashParam.ghostNormOffset[i] = in->GhostNormOffset(i) + in->Stride();
      dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
      ghost_threads[i] = in->GhostFace()[i] / 2;
    }

    void *gauge0, *gauge1;
    bindGaugeTex(gauge, parity, &gauge0, &gauge1);

    if (in->Precision() != gauge.Precision())
      errorQuda("Mixing gauge and spinor precision not supported");

    DslashCuda *dslash = 0;
    size_t regSize = sizeof(float);

    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
#if (__COMPUTE_CAPABILITY__ >= 130)
      dslash = new NdegTwistedDslashCuda<double2,double2>(out, (double2*)gauge0,(double2*)gauge1, gauge.Reconstruct(), in, x, type, kappa, mu, epsilon, k, dagger);
      regSize = sizeof(double);
#else
      errorQuda("Double precision not supported on this GPU");
#endif
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new NdegTwistedDslashCuda<float4,float4>(out, (float4*)gauge0,(float4*)gauge1, gauge.Reconstruct(), in, x, type, kappa, mu, epsilon, k, dagger);

    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      dslash = new NdegTwistedDslashCuda<short4,short4>(out, (short4*)gauge0,(short4*)gauge1, gauge.Reconstruct(), in, x, type, kappa, mu, epsilon, k, dagger);
    }

#ifndef GPU_COMMS
    DslashPolicyImp* dslashImp = DslashFactory::create(dslashPolicy);
#else
    DslashPolicyImp* dslashImp = DslashFactory::create(QUDA_GPU_COMMS_DSLASH);
#endif
    (*dslashImp)(*dslash, const_cast<cudaColorSpinorField*>(in), regSize, parity, dagger, bulk_threads, ghost_threads, profile);
    delete dslashImp;

    delete dslash;

    unbindGaugeTex(gauge);

    checkCudaError();
#else

#if (__COMPUTE_CAPABILITY__ < 200)
  errorQuda("Non-degenerate twisted-mass fermions not supported on pre-Fermi architecture");
#else
    errorQuda("Non-degenerate twisted mass dslash has not been built");
#endif

#endif
  }

}
