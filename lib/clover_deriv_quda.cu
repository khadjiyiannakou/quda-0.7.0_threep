#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <quda_internal.h>
#include <tune_quda.h>
#include <gauge_field.h>
#include <quda_matrix.h>
#include <cassert>

namespace quda {

#ifdef GPU_CLOVER_DIRAC

  template<class Cmplx>
    struct CloverDerivArg
    {
      int X[4];
      int border[4];
      int mu;
      int nu;
      typename RealTypeId<Cmplx>::Type coeff;
      int parity;
      int volumeCB;

      Cmplx* gauge;
      Cmplx* force;
      Cmplx* oprod;

      int forceStride;
      int gaugeStride;
      int oprodStride;

      int forceOffset;
      int gaugeOffset;
      int oprodOffset;
     
      bool conjugate;      

      CloverDerivArg(cudaGaugeField& force, cudaGaugeField& gauge, cudaGaugeField& oprod, int mu, int nu, double coeff, int parity, bool conjugate) :  
        mu(mu), nu(nu), coeff(coeff), parity(parity), volumeCB(force.VolumeCB()), 
        force(reinterpret_cast<Cmplx*>(force.Gauge_p())),  gauge(reinterpret_cast<Cmplx*>(gauge.Gauge_p())), oprod(reinterpret_cast<Cmplx*>(oprod.Gauge_p())),
        forceStride(force.Stride()), gaugeStride(gauge.Stride()), oprodStride(oprod.Stride()),
        forceOffset(force.Bytes()/(2*sizeof(Cmplx))), gaugeOffset(gauge.Bytes()/(2*sizeof(Cmplx))), oprodOffset(oprod.Bytes()/(2*sizeof(Cmplx)))
      {
        for(int dir=0; dir<4; ++dir) X[dir] = force.X()[dir];
        //for(int dir=0; dir<4; ++dir) border[dir] =  commDimPartitioned(dir) ? 2 : 0;
        for(int dir=0; dir<4; ++dir) border[dir] = 2;
      }
    };

  __device__ void getCoords(int x[4], int cb_index, const int X[4], int parity)
  {
    x[3] = cb_index/(X[2]*X[1]*X[0]/2);
    x[2] = (cb_index/(X[1]*X[0]/2)) % X[2];
    x[1] = (cb_index/(X[0]/2)) % X[1];
    x[0] = 2*(cb_index%(X[0]/2)) + ((x[3]+x[2]+x[1]+parity)&1);

    return;
  }

  __device__ int linkIndex(const int x[4], const int dx[4], const int X[4])
  {
    int y[4];
    for (int i=0; i<4; i++) y[i] = (x[i] + dx[i] + X[i]) % X[i];
    return (((y[3]*X[2] + y[2])*X[1] + y[1])*X[0] + y[0])/2;
  }


  template<typename Cmplx, bool isConjugate>
    __global__ void 
    cloverDerivativeKernel(const CloverDerivArg<Cmplx> arg)
    {
      int index = threadIdx.x + blockIdx.x*blockDim.x;

      if(index >= arg.volumeCB) return;


      int x[4];
      int y[4];
      int otherparity = (1-arg.parity);
      getCoords(x, index, arg.X, arg.parity);
      getCoords(y, index, arg.X, otherparity);
      int X[4]; 
      for(int dir=0; dir<4; ++dir) X[dir] = arg.X[dir];

      for(int dir=0; dir<4; ++dir){
        x[dir] += arg.border[dir];
        y[dir] += arg.border[dir];
        X[dir] += 2*arg.border[dir];
      }

      Cmplx* thisGauge = arg.gauge + arg.parity*arg.gaugeOffset;
      Cmplx* otherGauge = arg.gauge + (otherparity)*arg.gaugeOffset;

      Cmplx* thisOprod = arg.oprod + arg.parity*arg.oprodOffset;

      const int& mu = arg.mu;
      const int& nu = arg.nu;

      Matrix<Cmplx,3> thisForce;
      Matrix<Cmplx,3> otherForce;

      // U[mu](x) U[nu](x+mu) U[*mu](x+nu) U[*nu](x) Oprod(x)
      {
        int d[4] = {0, 0, 0, 0};

        // load U(x)_(+mu)
        Matrix<Cmplx,3> U1;
        loadLinkVariableFromArray(thisGauge, mu, linkIndex(x, d, X), 
            arg.gaugeStride, &U1);


        // load U(x+mu)_(+nu)
        Matrix<Cmplx,3> U2;
        d[mu]++;
        loadLinkVariableFromArray(otherGauge, nu, linkIndex(x, d, X), 
            arg.gaugeStride, &U2);
        d[mu]--;


        // load U(x+nu)_(+mu) 
        Matrix<Cmplx,3> U3;
        d[nu]++;
        loadLinkVariableFromArray(otherGauge, mu, linkIndex(x, d, X),
            arg.gaugeStride, &U3);
        d[nu]--;
      
        // load U(x)_(+nu)
        Matrix<Cmplx,3> U4;
        loadLinkVariableFromArray(thisGauge, nu, linkIndex(x, d, X),
            arg.gaugeStride, &U4);

        // load Oprod
        Matrix<Cmplx,3> Oprod1;
        loadMatrixFromArray(thisOprod, linkIndex(x, d, X), arg.oprodStride, &Oprod1);

        if(isConjugate) Oprod1 -= conj(Oprod1);
        thisForce = U1*U2*conj(U3)*conj(U4)*Oprod1;

        Matrix<Cmplx,3> Oprod2;
        d[mu]++; d[nu]++;
        loadMatrixFromArray(thisOprod, linkIndex(x, d, X), arg.oprodStride, &Oprod2);
        d[mu]--; d[nu]--;

        if(isConjugate) Oprod2 -= conj(Oprod2);

        thisForce += U1*U2*Oprod2*conj(U3)*conj(U4);

      } 
 
      { 
        int d[4] = {0, 0, 0, 0};
        // load U(x)_(+mu)
        Matrix<Cmplx,3> U1;
        loadLinkVariableFromArray(otherGauge, mu, linkIndex(y, d, X),
            arg.gaugeStride, &U1);

        // load U(x+mu)_(+nu)
        Matrix<Cmplx,3> U2;
        d[mu]++;
        loadLinkVariableFromArray(thisGauge, nu, linkIndex(y, d, X),
            arg.gaugeStride, &U2);
        d[mu]--;

        // load U(x+nu)_(+mu) 
        Matrix<Cmplx,3> U3;
        d[nu]++;
        loadLinkVariableFromArray(thisGauge, mu, linkIndex(y, d, X),
            arg.gaugeStride, &U3);
        d[nu]--;

        // load U(x)_(+nu)
        Matrix<Cmplx,3> U4;
        loadLinkVariableFromArray(otherGauge, nu, linkIndex(y, d, X),
            arg.gaugeStride, &U4);

        // load opposite parity Oprod
        Matrix<Cmplx,3> Oprod3;
        d[nu]++;
        loadMatrixFromArray(thisOprod, linkIndex(y, d, X), arg.oprodStride, &Oprod3);
        d[nu]--;

        if(isConjugate) Oprod3 -= conj(Oprod3);
        otherForce = U1*U2*conj(U3)*Oprod3*conj(U4);

        // load Oprod(x+mu)
        Matrix<Cmplx, 3> Oprod4;
        d[mu]++;
        loadMatrixFromArray(thisOprod, linkIndex(y, d, X), arg.oprodStride, &Oprod4);
        d[mu]--;

        if(isConjugate) Oprod4 -= conj(Oprod4);

        otherForce += U1*Oprod4*U2*conj(U3)*conj(U4);
      }


      // Lower leaf
      // U[nu*](x-nu) U[mu](x-nu) U[nu](x+mu-nu) Oprod(x+mu) U[*mu](x)
      {
        int d[4] = {0, 0, 0, 0};
        // load U(x-nu)(+nu)
        Matrix<Cmplx,3> U1;
        d[nu]--;
        loadLinkVariableFromArray(thisGauge, nu, linkIndex(y, d, X),
            arg.gaugeStride, &U1);
        d[nu]++;

        // load U(x-nu)(+mu) 
        Matrix<Cmplx, 3> U2;
        d[nu]--;
        loadLinkVariableFromArray(thisGauge, mu, linkIndex(y, d, X),
            arg.gaugeStride, &U2);
        d[nu]++;

        // load U(x+mu-nu)(nu)
        Matrix<Cmplx, 3> U3;
        d[mu]++; d[nu]--;
        loadLinkVariableFromArray(otherGauge, nu, linkIndex(y, d, X),
            arg.gaugeStride, &U3);
        d[mu]--; d[nu]++;

        // load U(x)_(+mu)
        Matrix<Cmplx,3> U4;
        loadLinkVariableFromArray(otherGauge, mu, linkIndex(y, d, X),
            arg.gaugeStride, &U4);

        // load Oprod(x+mu)
        Matrix<Cmplx, 3> Oprod1;
        d[mu]++;
        loadMatrixFromArray(thisOprod, linkIndex(y, d, X), arg.oprodStride, &Oprod1);
        d[mu]--;    

        if(isConjugate) Oprod1 -= conj(Oprod1);

        otherForce -= conj(U1)*U2*U3*Oprod1*conj(U4);

        Matrix<Cmplx,3> Oprod2;
        d[nu]--;
        loadMatrixFromArray(thisOprod, linkIndex(y, d, X), arg.oprodStride, &Oprod2);
        d[nu]++;

        if(isConjugate) Oprod2 -= conj(Oprod2);
        otherForce -= conj(U1)*Oprod2*U2*U3*conj(U4);
      }

      {
        int d[4] = {0, 0, 0, 0};
        // load U(x-nu)(+nu)
        Matrix<Cmplx,3> U1;
        d[nu]--;
        loadLinkVariableFromArray(otherGauge, nu, linkIndex(x, d, X), 
            arg.gaugeStride, &U1);
        d[nu]++;

        // load U(x-nu)(+mu) 
        Matrix<Cmplx, 3> U2;
        d[nu]--;
        loadLinkVariableFromArray(otherGauge, mu, linkIndex(x, d, X),
            arg.gaugeStride, &U2);
        d[nu]++;

        // load U(x+mu-nu)(nu)
        Matrix<Cmplx, 3> U3;
        d[mu]++; d[nu]--;
        loadLinkVariableFromArray(thisGauge, nu, linkIndex(x, d, X),
            arg.gaugeStride, &U3);
        d[mu]--; d[nu]++;

        // load U(x)_(+mu)
        Matrix<Cmplx,3> U4;
        loadLinkVariableFromArray(thisGauge, mu, linkIndex(x, d, X),
            arg.gaugeStride, &U4);


        Matrix<Cmplx,3> Oprod1;
        d[mu]++; d[nu]--;
        loadMatrixFromArray(thisOprod, linkIndex(x, d, X), arg.oprodStride, &Oprod1);
        d[mu]--; d[nu]++;

        if(isConjugate) Oprod1 -= conj(Oprod1);
        thisForce -= conj(U1)*U2*Oprod1*U3*conj(U4);

        Matrix<Cmplx, 3> Oprod4;
        loadMatrixFromArray(thisOprod, linkIndex(x, d, X), arg.oprodStride, &Oprod4);

        if(isConjugate) Oprod4 -= conj(Oprod4);
        thisForce -= Oprod4*conj(U1)*U2*U3*conj(U4);
      }
    
      thisForce *= arg.coeff;
      otherForce *= arg.coeff;


      // Write to array
      {
        appendMatrixToArray(thisForce, index, arg.forceStride, arg.force + arg.parity*arg.forceOffset);
        appendMatrixToArray(otherForce, index, arg.forceStride, arg.force + otherparity*arg.forceOffset); 
      }
      return;
    } // cloverDerivativeKernel


  template<typename Complex>
  class CloverDerivative : public Tunable {

  private:
    CloverDerivArg<Complex> arg;
    const GaugeField &meta;

    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &) const { return 0; }

    unsigned int minThreads() const { return arg.volumeCB; }
    bool tuneGridDim() const { return false; }

  public:
    CloverDerivative(const CloverDerivArg<Complex> &arg, const GaugeField &meta)
      : arg(arg), meta(meta) {
      writeAuxString("threads=%d,prec=%lu,stride=%d,geometery=%d",arg.volumeCB,sizeof(Complex)/2,arg.forceOffset);
    }
    virtual ~CloverDerivative() {}

    void apply(const hipStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if(arg.conjugate){
	cloverDerivativeKernel<Complex,true><<<tp.grid,tp.block,tp.shared_bytes>>>(arg);
      }else{
	cloverDerivativeKernel<Complex,false><<<tp.grid,tp.block,tp.shared_bytes>>>(arg);
      }
    } // apply

    void preTune(){}
    void postTune(){}

    long long flops() const {
      return 0;
    }

    long long bytes() const { return 0; }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }
  };


  // FIXME - the Tunable class isn't used here
  template<typename Float>
    void cloverDerivative(cudaGaugeField &out,
        cudaGaugeField& gauge,
        cudaGaugeField& oprod,
        int mu, int nu, double coeff, int parity,
        int conjugate)
    {
      typedef typename ComplexTypeId<Float>::Type Complex;
      CloverDerivArg<Complex> arg(out, gauge, oprod, mu, nu, coeff, parity, conjugate);
//      CloverDerivative<Complex> cloverDerivative(arg);
//      cloverDerivative.apply(0);
      dim3 blockDim(128, 1, 1);
      dim3 gridDim((arg.volumeCB + blockDim.x-1)/blockDim.x, 1, 1);
      if(conjugate){
        cloverDerivativeKernel<Complex,true><<<gridDim,blockDim,0>>>(arg);
      }else{
        cloverDerivativeKernel<Complex,false><<<gridDim,blockDim,0>>>(arg);
      }
    }    

#endif

  void cloverDerivative(cudaGaugeField &out,   
      cudaGaugeField& gauge,
      cudaGaugeField& oprod,
      int mu, int nu, double coeff, QudaParity parity, int conjugate)
  {
#ifdef GPU_CLOVER_DIRAC
    assert(oprod.Geometry() == QUDA_SCALAR_GEOMETRY);
    assert(out.Geometry() == QUDA_SCALAR_GEOMETRY);

    int device_parity = (parity == QUDA_EVEN_PARITY) ? 0 : 1;

    if(out.Precision() == QUDA_DOUBLE_PRECISION){
      cloverDerivative<double>(out, gauge, oprod, mu, nu, coeff, device_parity, conjugate);   
    } else if (out.Precision() == QUDA_SINGLE_PRECISION){
      cloverDerivative<float>(out, gauge, oprod, mu, nu, coeff, device_parity, conjugate);
    } else {
      errorQuda("Precision %d not supported", out.Precision());
    }
    return;
#else
    errorQuda("Clover has not been built");
#endif
  }              


} // namespace quda
