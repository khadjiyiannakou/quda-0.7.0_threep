#include "hip/hip_runtime.h"
#include <read_gauge.h>
#include <gauge_field.h>
#include <clover_field.h>
#include <dslash_quda.h>
#include <force_common.h>
#include <gauge_force_quda.h>
#ifdef MULTI_GPU
#include "face_quda.h"
#endif

namespace quda {

  namespace gaugeforce {
#include <dslash_constants.h>
#include <dslash_textures.h>
  } // namespace gaugeforce

  using namespace gaugeforce;


#define GF_SITE_MATRIX_LOAD_TEX 1

  //single precsison, 12-reconstruct
#if (GF_SITE_MATRIX_LOAD_TEX == 1)
#define LOAD_EVEN_MATRIX(dir, idx, var) LOAD_MATRIX_12_SINGLE_TEX(siteLink0TexSingle_recon, dir, idx, var, gf.site_ga_stride)
#define LOAD_ODD_MATRIX(dir, idx, var) 	LOAD_MATRIX_12_SINGLE_TEX(siteLink1TexSingle_recon, dir, idx, var, gf.site_ga_stride)
#else
#define LOAD_EVEN_MATRIX(dir, idx, var) LOAD_MATRIX_12_SINGLE(linkEven, dir, idx, var, gf.site_ga_stride)
#define LOAD_ODD_MATRIX(dir, idx, var) LOAD_MATRIX_12_SINGLE(linkOdd, dir, idx, var, gf.site_ga_stride)
#endif
#define LOAD_ANTI_HERMITIAN(src, dir, idx, var) LOAD_ANTI_HERMITIAN_DIRECT(src, dir, idx, var, gf.mom_ga_stride)
#define RECONSTRUCT_MATRIX(sign, var) RECONSTRUCT_LINK_12(sign,var)
#define DECLARE_LINK_VARS(var) FloatN var##0, var##1, var##2, var##3, var##4
#define N_IN_FLOATN 4
#define GAUGE_FORCE_KERN_NAME parity_compute_gauge_force_kernel_sp12
#include "gauge_force_core.h"
#undef LOAD_EVEN_MATRIX 
#undef LOAD_ODD_MATRIX
#undef LOAD_ANTI_HERMITIAN 
#undef RECONSTRUCT_MATRIX
#undef DECLARE_LINK_VARS
#undef N_IN_FLOATN 
#undef GAUGE_FORCE_KERN_NAME

  //double precsison, 12-reconstruct
#if (GF_SITE_MATRIX_LOAD_TEX == 1)
#define LOAD_EVEN_MATRIX(dir, idx, var) LOAD_MATRIX_12_DOUBLE_TEX(siteLink0TexDouble, linkEven, dir, idx, var, gf.site_ga_stride)
#define LOAD_ODD_MATRIX(dir, idx, var) 	LOAD_MATRIX_12_DOUBLE_TEX(siteLink1TexDouble, linkOdd, dir, idx, var, gf.site_ga_stride)
#else
#define LOAD_EVEN_MATRIX(dir, idx, var) LOAD_MATRIX_12_DOUBLE(linkEven, dir, idx, var, gf.site_ga_stride)
#define LOAD_ODD_MATRIX(dir, idx, var) LOAD_MATRIX_12_DOUBLE(linkOdd, dir, idx, var, gf.site_ga_stride)
#endif
#define LOAD_ANTI_HERMITIAN(src, dir, idx, var) LOAD_ANTI_HERMITIAN_DIRECT(src, dir, idx, var, gf.mom_ga_stride)
#define RECONSTRUCT_MATRIX(sign, var) RECONSTRUCT_LINK_12(sign,var)
#define DECLARE_LINK_VARS(var) FloatN var##0, var##1, var##2, var##3, var##4, var##5, var##6, var##7, var##8 
#define N_IN_FLOATN 2
#define GAUGE_FORCE_KERN_NAME parity_compute_gauge_force_kernel_dp12
#include "gauge_force_core.h"
#undef LOAD_EVEN_MATRIX 
#undef LOAD_ODD_MATRIX
#undef LOAD_ANTI_HERMITIAN 
#undef RECONSTRUCT_MATRIX
#undef DECLARE_LINK_VARS
#undef N_IN_FLOATN 
#undef GAUGE_FORCE_KERN_NAME

  //single precision, 18-reconstruct
#if (GF_SITE_MATRIX_LOAD_TEX == 1)
#define LOAD_EVEN_MATRIX(dir, idx, var) LOAD_MATRIX_18_SINGLE_TEX(siteLink0TexSingle, dir, idx, var, gf.site_ga_stride)
#define LOAD_ODD_MATRIX(dir, idx, var) 	LOAD_MATRIX_18_SINGLE_TEX(siteLink1TexSingle, dir, idx, var, gf.site_ga_stride)
#else
#define LOAD_EVEN_MATRIX(dir, idx, var) LOAD_MATRIX_18(linkEven, dir, idx, var, gf.site_ga_stride)
#define LOAD_ODD_MATRIX(dir, idx, var) LOAD_MATRIX_18(linkOdd, dir, idx, var, gf.site_ga_stride)
#endif
#define LOAD_ANTI_HERMITIAN(src, dir, idx, var) LOAD_ANTI_HERMITIAN_DIRECT(src, dir, idx, var,gf.mom_ga_stride)
#define RECONSTRUCT_MATRIX(sign, var) 
#define DECLARE_LINK_VARS(var) FloatN var##0, var##1, var##2, var##3, var##4, var##5, var##6, var##7, var##8 
#define N_IN_FLOATN 2
#define GAUGE_FORCE_KERN_NAME parity_compute_gauge_force_kernel_sp18
#include "gauge_force_core.h"
#undef LOAD_EVEN_MATRIX
#undef LOAD_ODD_MATRIX
#undef LOAD_ANTI_HERMITIAN 
#undef RECONSTRUCT_MATRIX
#undef DECLARE_LINK_VARS
#undef N_IN_FLOATN 
#undef GAUGE_FORCE_KERN_NAME

  //double precision, 18-reconstruct
#if (GF_SITE_MATRIX_LOAD_TEX == 1)
#define LOAD_EVEN_MATRIX(dir, idx, var) LOAD_MATRIX_18_DOUBLE_TEX(siteLink0TexDouble, linkEven, dir, idx, var, gf.site_ga_stride)
#define LOAD_ODD_MATRIX(dir, idx, var) 	LOAD_MATRIX_18_DOUBLE_TEX(siteLink1TexDouble, linkOdd, dir, idx, var, gf.site_ga_stride)
#else
#define LOAD_EVEN_MATRIX(dir, idx, var) LOAD_MATRIX_18(linkEven, dir, idx, var, gf.site_ga_stride)
#define LOAD_ODD_MATRIX(dir, idx, var) LOAD_MATRIX_18(linkOdd, dir, idx, var, gf.site_ga_stride)
#endif
#define LOAD_ANTI_HERMITIAN(src, dir, idx, var) LOAD_ANTI_HERMITIAN_DIRECT(src, dir, idx, var, gf.mom_ga_stride)
#define RECONSTRUCT_MATRIX(sign, var) 
#define DECLARE_LINK_VARS(var) FloatN var##0, var##1, var##2, var##3, var##4, var##5, var##6, var##7, var##8 
#define N_IN_FLOATN 2
#define GAUGE_FORCE_KERN_NAME parity_compute_gauge_force_kernel_dp18
#include "gauge_force_core.h"
#undef LOAD_EVEN_MATRIX
#undef LOAD_ODD_MATRIX
#undef LOAD_ANTI_HERMITIAN 
#undef RECONSTRUCT_MATRIX
#undef DECLARE_LINK_VARS
#undef N_IN_FLOATN 
#undef GAUGE_FORCE_KERN_NAME

  void
  gauge_force_init_cuda(QudaGaugeParam* param, int path_max_length)
  {    
  
    static int gauge_force_init_cuda_flag = 0;
    if (gauge_force_init_cuda_flag){
      return;
    }
    gauge_force_init_cuda_flag=1;

    int* X = param->X;
  
    int Vh = X[0]*X[1]*X[2]*X[3]/2;
    fat_force_const_t gf_h;
    gf_h.path_max_length = path_max_length;  
#ifdef MULTI_GPU  
    int Vh_ex = (X[0]+4)*(X[1]+4)*(X[2]+4)*(X[3]+4)/2; // FIXME - this should not be hardcoded
    gf_h.site_ga_stride = param->site_ga_pad + Vh_ex;
#else  
    gf_h.site_ga_stride = param->site_ga_pad + Vh;
#endif
  
    gf_h.mom_ga_stride = param->mom_ga_pad + Vh;  
    hipMemcpyToSymbol(HIP_SYMBOL(gf), &gf_h, sizeof(fat_force_const_t));     
  }


  class GaugeForceCuda : public Tunable {

  private:
    cudaGaugeField &mom;
    const int dir;
    const double eb3;
    const cudaGaugeField &link;
    const int *input_path;
    const int *length;
    const double *path_coeff;
    const int num_paths;
    const kernel_param_t &kparam;

    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &) const { return 0; }
  
    // don't tune the grid dimension
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return kparam.threads; }

  public:
    GaugeForceCuda(cudaGaugeField &mom, const int dir, const double &eb3, const cudaGaugeField &link,
		   const int *input_path, const int *length, const double *path_coeff, 
		   const int num_paths, const kernel_param_t &kparam) :
      mom(mom), dir(dir), eb3(eb3), link(link), input_path(input_path), length(length), 
      path_coeff(path_coeff), num_paths(num_paths), kparam(kparam) { 

      if(link.Precision() == QUDA_DOUBLE_PRECISION){
	hipBindTexture(0, siteLink0TexDouble, link.Even_p(), link.Bytes()/2);
	hipBindTexture(0, siteLink1TexDouble, link.Odd_p(), link.Bytes()/2);			      
      }else{ //QUDA_SINGLE_PRECISION
	if(link.Reconstruct() == QUDA_RECONSTRUCT_NO){
	  hipBindTexture(0, siteLink0TexSingle, link.Even_p(), link.Bytes()/2);
	  hipBindTexture(0, siteLink1TexSingle, link.Odd_p(), link.Bytes()/2);		
	}else{//QUDA_RECONSTRUCT_12
	  hipBindTexture(0, siteLink0TexSingle_recon, link.Even_p(), link.Bytes()/2);
	  hipBindTexture(0, siteLink1TexSingle_recon, link.Odd_p(), link.Bytes()/2);	
	}
      }
    }

    virtual ~GaugeForceCuda() {
      if(link.Precision() == QUDA_DOUBLE_PRECISION){
	hipBindTexture(0, siteLink0TexDouble, link.Even_p(), link.Bytes()/2);
	hipBindTexture(0, siteLink1TexDouble, link.Odd_p(), link.Bytes()/2);			      
      }else{ //QUDA_SINGLE_PRECISION
	if(link.Reconstruct() == QUDA_RECONSTRUCT_NO){
	  hipBindTexture(0, siteLink0TexSingle, link.Even_p(), link.Bytes()/2);
	  hipBindTexture(0, siteLink1TexSingle, link.Odd_p(), link.Bytes()/2);		
	}else{//QUDA_RECONSTRUCT_12
	  hipBindTexture(0, siteLink0TexSingle_recon, link.Even_p(), link.Bytes()/2);
	  hipBindTexture(0, siteLink1TexSingle_recon, link.Odd_p(), link.Bytes()/2);	
	}
      }
    }

    void apply(const hipStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());    
      if(link.Precision() == QUDA_DOUBLE_PRECISION){      
	if(link.Reconstruct() == QUDA_RECONSTRUCT_NO){
	  parity_compute_gauge_force_kernel_dp18<0,double><<<tp.grid, tp.block>>>((double2*)mom.Even_p(), (double2*)mom.Odd_p(),
									   dir, eb3,
									   (double2*)link.Even_p(), (double2*)link.Odd_p(), 
									   input_path, length, path_coeff,
									   num_paths, kparam);   
	  parity_compute_gauge_force_kernel_dp18<1,double><<<tp.grid, tp.block>>>((double2*)mom.Even_p(), (double2*)mom.Odd_p(),
									   dir, eb3,
									   (double2*)link.Even_p(), (double2*)link.Odd_p(), 
									   input_path, length, path_coeff,
									   num_paths, kparam);  
	
	}else{ //QUDA_RECONSTRUCT_12
	  parity_compute_gauge_force_kernel_dp12<0,double><<<tp.grid, tp.block>>>((double2*)mom.Even_p(), (double2*)mom.Odd_p(),
									   dir, eb3,
									   (double2*)link.Even_p(), (double2*)link.Odd_p(), 
									   input_path, length, path_coeff,
									   num_paths, kparam);   
	  parity_compute_gauge_force_kernel_dp12<1,double><<<tp.grid, tp.block>>>((double2*)mom.Even_p(), (double2*)mom.Odd_p(),
									   dir, eb3,
									   (double2*)link.Even_p(), (double2*)link.Odd_p(), 
									   input_path, length, path_coeff,
									   num_paths, kparam);    
	}
      }else{ //QUDA_SINGLE_PRECISION
	if(link.Reconstruct() == QUDA_RECONSTRUCT_NO){
	
	  parity_compute_gauge_force_kernel_sp18<0,float><<<tp.grid, tp.block>>>((float2*)mom.Even_p(), (float2*)mom.Odd_p(),
									   dir, eb3,
									   (float2*)link.Even_p(), (float2*)link.Odd_p(), 
									   input_path, length, path_coeff,
									   num_paths, kparam);   
	  parity_compute_gauge_force_kernel_sp18<1,float><<<tp.grid, tp.block>>>((float2*)mom.Even_p(), (float2*)mom.Odd_p(),
									   dir, eb3,
									   (float2*)link.Even_p(), (float2*)link.Odd_p(), 
									   input_path, length, path_coeff,
									   num_paths, kparam); 
	
	}else{ //QUDA_RECONSTRUCT_12
	  parity_compute_gauge_force_kernel_sp12<0,float><<<tp.grid, tp.block>>>((float2*)mom.Even_p(), (float2*)mom.Odd_p(),
									   dir, eb3,
									   (float4*)link.Even_p(), (float4*)link.Odd_p(), 
									   input_path, length, path_coeff,
									   num_paths, kparam);   
	  //odd
	  /* The reason we do not switch the even/odd function input paramemters and the texture binding
	   * is that we use the oddbit to decided where to load, in the kernel function
	   */
	  parity_compute_gauge_force_kernel_sp12<1,float><<<tp.grid, tp.block>>>((float2*)mom.Even_p(), (float2*)mom.Odd_p(),
									   dir, eb3,
									   (float4*)link.Even_p(), (float4*)link.Odd_p(), 
									   input_path, length, path_coeff,
									   num_paths, kparam);  
	}
      }
    }
  
    void preTune() { mom.backup(); }
    void postTune() { mom.restore(); } 
  
    long long flops() const { return 0; } // FIXME: add flops counter
  
    TuneKey tuneKey() const {
      std::stringstream vol, aux;
      vol << link.X()[0] << "x";
      vol << link.X()[1] << "x";
      vol << link.X()[2] << "x";
      vol << link.X()[3];
      aux << "threads=" << link.Volume() << ",prec=" << link.Precision();
      aux << "stride=" << link.Stride() << ",recon=" << link.Reconstruct();
      aux << "dir=" << dir << "num_paths=" << num_paths;
      return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
    }  
  
  };
  
  void
  gauge_force_cuda_dir(cudaGaugeField& cudaMom, const int dir, const double eb3, const cudaGaugeField& cudaSiteLink,
		       const QudaGaugeParam* param, int** input_path, const int* length, const double* path_coeff, 
		       const int num_paths, const int max_length)
  {
    //input_path
    size_t bytes = num_paths*max_length*sizeof(int);

    int *input_path_d = (int *) device_malloc(bytes);
    hipMemset(input_path_d, 0, bytes);
    checkCudaError();

    int* input_path_h = (int *) safe_malloc(bytes);
    memset(input_path_h, 0, bytes);

    for(int i=0; i < num_paths; i++) {
      for(int j=0; j < length[i]; j++) {
	input_path_h[i*max_length + j] = input_path[i][j];
      }
    }

    hipMemcpy(input_path_d, input_path_h, bytes, hipMemcpyHostToDevice); 
    
    //length
    int* length_d = (int *) device_malloc(num_paths*sizeof(int));
    hipMemcpy(length_d, length, num_paths*sizeof(int), hipMemcpyHostToDevice);
    
    //path_coeff
    void* path_coeff_d = device_malloc(num_paths*sizeof(double));
    hipMemcpy(path_coeff_d, path_coeff, num_paths*sizeof(double), hipMemcpyHostToDevice); 

    //compute the gauge forces
    int volume = param->X[0]*param->X[1]*param->X[2]*param->X[3];
        
    kernel_param_t kparam;
#ifdef MULTI_GPU
    for(int i=0; i<4; i++) {
      kparam.ghostDim[i] = commDimPartitioned(i);
    }
#endif
    kparam.threads = volume/2;

    GaugeForceCuda gaugeForce(cudaMom, dir, eb3, cudaSiteLink, input_path_d, 
			      length_d, reinterpret_cast<double*>(path_coeff_d), num_paths, kparam);
    gaugeForce.apply(0);
    checkCudaError();
    
    host_free(input_path_h);
    device_free(input_path_d);
    device_free(length_d);
    device_free(path_coeff_d);
  }


  void
  gauge_force_cuda(cudaGaugeField&  cudaMom, double eb3, cudaGaugeField& cudaSiteLink,
		   QudaGaugeParam* param, int*** input_path, 
		   int* length, double* path_coeff, int num_paths, int max_length)
  {  
    for(int dir=0; dir < 4; dir++){
      gauge_force_cuda_dir(cudaMom, dir, eb3, cudaSiteLink, param, input_path[dir], 
			   length, path_coeff, num_paths, max_length);
    }  
  }

} // namespace quda
